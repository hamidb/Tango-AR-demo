#include "hip/hip_runtime.h"
#include "nativeCUDA.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define DEBUG

inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG)
	if (result != hipSuccess) {
		LOGI("CUDA Runtime Error: %sn", hipGetErrorString(result));
	}
#endif
	return result;
}

__global__
void greyKernel(uchar* d_input, uchar* d_output, int rows, int cols);

void launchGreyKernel(uchar* d_input, uchar* d_output, int rows, int cols) {
	const dim3 blockSize(TPB, TPB, 1); 
	const dim3 gridSize( (cols + TPB - 1)/TPB, (rows + TPB - 1)/TPB, 1);
    greyKernel<<<gridSize, blockSize>>>(d_input, d_output, rows, cols);
}

void CUDA_greyCvt(uchar* input, uchar** output, int rows, int cols) {

    size_t pixel_size = rows * cols;

    // Allocate device space
    uchar *d_input, *d_output;
    checkCuda (hipMalloc((void**) &d_input, 3 * pixel_size) );
    checkCuda (hipMalloc((void**) &d_output, pixel_size) );
	checkCuda (hipMemset((void*) d_output, 0, pixel_size));

    // Copy input image to device memory asynchronously
    checkCuda( hipMemcpyAsync(d_input, input, 3 * pixel_size, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpyAsync(d_output, *output, pixel_size, hipMemcpyHostToDevice) );

    // Wait for copies to complete
    hipDeviceSynchronize();

    // Launch device kernel
    launchGreyKernel(d_input, d_output, rows, cols);

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Check for any errors created by kernel
    checkCuda(hipGetLastError());

    // Copy back sum array
    checkCuda( hipMemcpy(*output, d_output, pixel_size, hipMemcpyDeviceToHost) );

    // Free allocated memory
    hipFree(d_input);
    hipFree(d_output);
}

// GPU kernel
__global__ 
void greyKernel(uchar* d_input, uchar* d_output, int rows, int cols){
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
	int index_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (index_x >= cols || index_y >= rows) {
        return;
    }

	uchar* rgb = &d_input[3*(index_x + index_y * cols)];
    d_output[index_x + index_y*cols] = rgb[0] * .299f + rgb[1] * .587f + rgb[2] * .114f;
}
